#include "hip/hip_runtime.h"
#include "app.cuh"

// static __global__ void sample_kernel(Sampler_new *sampler) {
//   Jobs_result<JobType::NS, uint> &result = sampler->result;
//   gpu_graph *graph = &sampler->ggraph;
//   hiprandState state;
//   hiprand_init(TID, 0, 0, &state);
//   __shared__ uint current_itr;
//   if (threadIdx.x == 0) current_itr = 0;
//   __syncthreads();

//   for (; current_itr < result.hop_num - 1;)  // for 2-hop, hop_num=3
//   {
//     sample_job job;
//     __threadfence_block();
//     job = result.requireOneJob(current_itr);
//     while (job.val && graph->CheckValid(job.node_id)) {
//       uint src_id = job.node_id;
//       Vector_virtual<uint> alias;
//       Vector_virtual<float> prob;
//       uint src_degree = graph->getDegree((uint)src_id);
//       alias.Construt(
//           graph->alias_array + graph->xadj[src_id] - graph->local_vtx_offset,
//           src_degree);
//       prob.Construt(
//           graph->prob_array + graph->xadj[src_id] - graph->local_vtx_offset,
//           src_degree);
//       alias.Init(src_degree);
//       prob.Init(src_degree);
//       {
//         uint target_size = result.hops[current_itr + 1];
//         if ((target_size > 0) && (target_size < src_degree)) {
//           //   int itr = 0;
//           for (size_t i = 0; i < target_size; i++) {
//             int col = (int)floor(hiprand_uniform(&state) * src_degree);
//             float p = hiprand_uniform(&state);
//             uint candidate;
//             if (p < prob[col])
//               candidate = col;
//             else
//               candidate = alias[col];
//             result.AddActive(current_itr, result.getNextAddr(current_itr),
//                              graph->getOutNode(src_id, candidate));
//           }
//         } else if (target_size >= src_degree) {
//           for (size_t i = 0; i < src_degree; i++) {
//             result.AddActive(current_itr, result.getNextAddr(current_itr),
//                              graph->getOutNode(src_id, i));
//           }
//         }
//       }

//       job = result.requireOneJob(current_itr);
//     }
//     __syncthreads();
//     if (threadIdx.x == 0) result.NextItr(current_itr);
//     __syncthreads();
//   }
// }

static __global__ void sample_kernel_first(Sampler_new *sampler, uint itr) {
  Jobs_result<JobType::NS, uint> &result = sampler->result;
  gpu_graph *graph = &sampler->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);
  __shared__ matrixBuffer<BLOCK_SIZE, 10, uint> buffer_1hop;
  Vector_virtual<uint> alias;
  Vector_virtual<float> prob;

  buffer_1hop.Init();
  size_t idx_i = TID;
  if (idx_i < result.size) {
    uint current_itr = 0;
    coalesced_group active = coalesced_threads();
    {
      uint src_id = result.GetData(idx_i, current_itr, 0);
      uint src_degree = graph->getDegree((uint)src_id);
      uint sample_size = MIN(result.hops[current_itr + 1], src_degree);

      alias.Construt(
          graph->alias_array + graph->xadj[src_id] - graph->local_vtx_offset,
          src_degree);
      prob.Construt(
          graph->prob_array + graph->xadj[src_id] - graph->local_vtx_offset,
          src_degree);
      alias.Init(src_degree);
      prob.Init(src_degree);

      for (size_t i = 0; i < sample_size; i++) {
        int col = (int)floor(hiprand_uniform(&state) * src_degree);
        float p = hiprand_uniform(&state);
        uint candidate;
        if (p < prob[col])
          candidate = col;
        else
          candidate = alias[col];

        // *result.GetDataPtr(idx_i, current_itr + 1, i) =
        //       graph->getOutNode(src_id, candidate);
        buffer_1hop.Set(
            graph->getOutNode(src_id, candidate));  // can move back latter
      }
      active.sync();
      buffer_1hop.Flush(result.data + result.length_per_sample * idx_i, 0);
      result.SetSampleLength(idx_i, current_itr, 0, sample_size);
    }
  }
}
template <uint subwarp_size>
static __global__ void sample_kernel_second_buffer(Sampler_new *sampler,
                                                   uint current_itr) {
#define buffer_len 15  // occupancy allows 15, 15 75% occupancy but best?
  Jobs_result<JobType::NS, uint> &result = sampler->result;
  gpu_graph *graph = &sampler->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  size_t subwarp_id = TID / subwarp_size;
  uint subwarp_idx = TID % subwarp_size;
  uint local_subwarp_id = LTID / subwarp_size;
  bool alive = (subwarp_idx < result.hops[current_itr]) ? 1 : 0;
  size_t idx_i = subwarp_id;  //

  Vector_virtual<uint> alias;
  Vector_virtual<float> prob;

  thread_block tb = this_thread_block();
  auto warp = tiled_partition<32>(tb);
  auto subwarp = tiled_partition<subwarp_size>(warp);

  __shared__ uint buffer[BLOCK_SIZE][buffer_len];
  // buffer.Init();
  __shared__ uint idxMap[BLOCK_SIZE];
  __shared__ uint iMap[BLOCK_SIZE];
  __shared__ uint len[BLOCK_SIZE];
  // __shared__ uint MainLen[BLOCK_SIZE / subwarp_size];
  idxMap[LTID] = 0;
  iMap[LTID] = 0;
  len[LTID] = 0;
  // if (!subwarp.thread_rank()) MainLen[LTID] = 0;

  if (idx_i < result.size)  // for 2-hop, hop_num=3
  {
    idxMap[LTID] = idx_i;
    iMap[LTID] = subwarp_idx;
    coalesced_group active = coalesced_threads();
    {
      uint src_id, sample_size, src_degree = 0;
      if (alive) {
        src_id = result.GetData(idx_i, current_itr, subwarp_idx);
        src_degree = graph->getDegree((uint)src_id);
        alive = (src_degree == 0) ? false : true;
      }
      // sample_size = MIN(result.hops[current_itr + 1], src_degree);
      sample_size = result.hops[current_itr + 1];
      alias.Construt(
          graph->alias_array + graph->xadj[src_id] - graph->local_vtx_offset,
          src_degree);
      prob.Construt(
          graph->prob_array + graph->xadj[src_id] - graph->local_vtx_offset,
          src_degree);
      alias.Init(src_degree);
      prob.Init(src_degree);

      for (size_t i = 0; i < sample_size; i++) {
        if (alive) {
          // uint candidate = (int)floor(hiprand_uniform(&state) * src_degree);
          // *result.GetDataPtr(idx_i, current_itr + 1, i) =
          //     graph->getOutNode(src_id, candidate);
          int col = (int)floor(hiprand_uniform(&state) * src_degree);
          float p = hiprand_uniform(&state);
          uint candidate;
          if (p < prob[col])
            candidate = col;
          else
            candidate = alias[col];
          buffer[LTID][len[LTID]] = graph->getOutNode(src_id, candidate);
          len[LTID] += 1;
        }
        subwarp.sync();
        uint mainLen = cg::reduce(subwarp, len[LTID], cg::greater<uint>());
        if (mainLen == buffer_len) {
          for (size_t j = 0; j < subwarp_size; j++) {
            subwarp.sync();
            for (size_t k = subwarp.thread_rank();
                 k < len[local_subwarp_id * subwarp_size + j];
                 k += subwarp.size()) {
              *result.GetDataPtr(idxMap[local_subwarp_id * subwarp_size + j],
                                 current_itr + 1, k) =
                  buffer[local_subwarp_id * subwarp_size + j][k];
            }
            if (subwarp.thread_rank() == 0)
              len[local_subwarp_id * subwarp_size + j] = 0;
          }
        }
      }

      if (alive)
        result.SetSampleLength(idx_i, current_itr, subwarp_idx, sample_size);
      subwarp.sync();
      for (size_t j = 0; j < subwarp_size; j++) {
        subwarp.sync();
        for (size_t k = subwarp.thread_rank();
             k < len[local_subwarp_id * subwarp_size + j];
             k += subwarp.size()) {
          *result.GetDataPtr(idxMap[local_subwarp_id * subwarp_size + j],
                             current_itr + 1, k) =
              buffer[local_subwarp_id * subwarp_size + j][k];
        }
      }
    }
  }
}
template <uint subwarp_size>
static __global__ void sample_kernel_second(Sampler_new *sampler,
                                            uint current_itr) {
  Jobs_result<JobType::NS, uint> &result = sampler->result;
  gpu_graph *graph = &sampler->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);
  size_t subwarp_id = TID / subwarp_size;
  uint subwarp_idx = TID % subwarp_size;
  uint local_subwarp_id = LTID % subwarp_size;
  bool alive = (subwarp_idx < result.hops[current_itr]) ? 1 : 0;
  size_t idx_i = subwarp_id;  //
  Vector_virtual<uint> alias;
  Vector_virtual<float> prob;

  if (idx_i < result.size)  // for 2-hop, hop_num=3
  {
    coalesced_group active = coalesced_threads();
    {
      uint src_id, src_degree, sample_size;
      if (alive) {
        src_id = result.GetData(idx_i, current_itr, subwarp_idx);
        src_degree = graph->getDegree((uint)src_id);
        sample_size = MIN(result.hops[current_itr + 1], src_degree);
        alias.Construt(
            graph->alias_array + graph->xadj[src_id] - graph->local_vtx_offset,
            src_degree);
        prob.Construt(
            graph->prob_array + graph->xadj[src_id] - graph->local_vtx_offset,
            src_degree);
        alias.Init(src_degree);
        prob.Init(src_degree);
        for (size_t i = 0; i < sample_size; i++) {
          int col = (int)floor(hiprand_uniform(&state) * src_degree);
          float p = hiprand_uniform(&state);
          uint candidate;
          if (p < prob[col])
            candidate = col;
          else
            candidate = alias[col];
          *result.GetDataPtr(idx_i, current_itr + 1, i) =
              graph->getOutNode(src_id, candidate);
        }
      }
      if (alive)
        result.SetSampleLength(idx_i, current_itr, subwarp_idx, sample_size);
    }
  }
}

static __global__ void print_result(Sampler_new *sampler) {
  sampler->result.PrintResult();
}

float OfflineSample(Sampler_new &sampler) {
  LOG("%s\n", __FUNCTION__);
  int device;
  hipDeviceProp_t prop;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  int n_sm = prop.multiProcessorCount;

  Sampler_new *sampler_ptr;
  hipMalloc(&sampler_ptr, sizeof(Sampler_new));
  CUDA_RT_CALL(hipMemcpy(sampler_ptr, &sampler, sizeof(Sampler_new),
                          hipMemcpyHostToDevice));
  double start_time, total_time;
  //   init_kernel_ptr<<<1, 32, 0, 0>>>(sampler_ptr, true);

  // allocate global buffer
  int block_num = n_sm * FLAGS_m;

  CUDA_RT_CALL(hipDeviceSynchronize());
  CUDA_RT_CALL(hipPeekAtLastError());
  start_time = wtime();
  sample_kernel_first<<<sampler.result.size / BLOCK_SIZE + 1, BLOCK_SIZE, 0,
                        0>>>(sampler_ptr, 0);
  sample_kernel_second<16>
      <<<sampler.result.size * 16 / BLOCK_SIZE + 1, BLOCK_SIZE, 0, 0>>>(
          sampler_ptr, 1);
  CUDA_RT_CALL(hipDeviceSynchronize());
  // CUDA_RT_CALL(hipPeekAtLastError());
  total_time = wtime() - start_time;
  LOG("Device %d sampling time:\t%.2f ms ratio:\t %.1f MSEPS\n",
      omp_get_thread_num(), total_time * 1000,
      static_cast<float>(sampler.result.GetSampledNumber() / total_time /
                         1000000));
  sampler.sampled_edges = sampler.result.GetSampledNumber();
  LOG("sampled_edges %d\n", sampler.sampled_edges);
  if (FLAGS_printresult) print_result<<<1, 32, 0, 0>>>(sampler_ptr);
  CUDA_RT_CALL(hipDeviceSynchronize());
  return total_time;
}
