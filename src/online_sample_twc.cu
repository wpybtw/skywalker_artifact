#include "hip/hip_runtime.h"
#include "app.cuh"

using block_table =
    alias_table_constructor_shmem<uint, thread_block, BufferType::GMEM>;
using warp_table = alias_table_constructor_shmem<uint, thread_block_tile<32>>;
using subwarp_table =
    alias_table_constructor_shmem<uint, thread_block_tile<SUBWARP_SIZE>,
                                  BufferType::SHMEM,
                                  AliasTableStorePolicy::NONE>;
                                  
static __device__ void SampleSubwarpCentic(sample_result &result,
                                           gpu_graph *ggraph, hiprandState state,
                                           int current_itr, int idx,
                                           int node_id, void *buffer) {
  subwarp_table *tables = (subwarp_table *)buffer;
  subwarp_table *table = &tables[SWID];
  bool not_all_zero =
      table->loadFromGraph(ggraph->getNeighborPtr(node_id), ggraph,
                           ggraph->getDegree(node_id), current_itr, node_id);
  if (not_all_zero) {
    table->construct();
    table->roll_atomic(&state, result);
  }
  table->Clean();
}

static __device__ void SampleWarpCentic(sample_result &result,
                                        gpu_graph *ggraph, hiprandState state,
                                        int current_itr, int idx, int node_id,
                                        void *buffer) {
  // subwarp_table *tables = (subwarp_table *)buffer;
  warp_table *tables =
      (warp_table *)((void *)buffer +
                     WID * WARP_SIZE / SUBWARP_SIZE * sizeof(subwarp_table));
  warp_table *table = &tables[0];
  bool not_all_zero =
      table->loadFromGraph(ggraph->getNeighborPtr(node_id), ggraph,
                           ggraph->getDegree(node_id), current_itr, node_id);
  if (not_all_zero) {
    table->construct();
    table->roll_atomic(&state, result);
  }
  table->Clean();
}

static __device__ void SampleBlockCentic(sample_result &result,
                                         gpu_graph *ggraph, hiprandState state,
                                         int current_itr, int node_id,
                                         void *buffer,
                                         Vector_pack<uint> *vector_packs) {
  block_table *tables = (block_table *)buffer;
  block_table *table = &tables[0];
  table->loadGlobalBuffer(vector_packs);
  __syncthreads();
  bool not_all_zero =
      table->loadFromGraph(ggraph->getNeighborPtr(node_id), ggraph,
                           ggraph->getDegree(node_id), current_itr, node_id);
  __syncthreads();
  if (not_all_zero) {
    table->constructBC();
    uint target_size =
        MIN(ggraph->getDegree(node_id), result.hops[current_itr + 1]);
    table->roll_atomic(target_size, &state, result);
  }
  __syncthreads();
  table->Clean();
}

// template <typename T>
// struct worker;

static __global__ void sample_kernel(Sampler *sampler,
                                     Vector_pack<uint> *vector_pack) {
  sample_result &result = sampler->result;
  gpu_graph *ggraph = &sampler->ggraph;
  Vector_pack<uint> *vector_packs = &vector_pack[BID];
  __shared__ subwarp_table table[SUBWARP_PER_BLK];

  void *buffer = &table[0];
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  thread_block tb = this_thread_block();
  auto warp = tiled_partition<32>(tb);
  auto subwarp = tiled_partition<4>(warp);

  __shared__ uint current_itr;
  if (threadIdx.x == 0) current_itr = 0;
  __syncthreads();
  for (; current_itr < result.hop_num - 1;)  // for 2-hop, hop_num=3
  {
    sample_job job;
    __threadfence_block();

    if (subwarp.thread_rank() == 0) {
      job = result.requireOneJob(current_itr);
    }
    subwarp.sync();
    job.idx = subwarp.shfl(job.idx, 0);
    job.val = subwarp.shfl(job.val, 0);
    job.node_id = subwarp.shfl(job.node_id, 0);
    subwarp.sync();
    while (job.val) {
      subwarp.sync();
      if (ggraph->getDegree(job.node_id) < ELE_PER_SUBWARP) {
        SampleSubwarpCentic(result, ggraph, state, current_itr, job.idx,
                            job.node_id, buffer);
      } else if (ggraph->getDegree(job.node_id) < ELE_PER_WARP) {
        if (subwarp.thread_rank() == 0) {
          result.AddMidDegree(current_itr, job.node_id);
        }
      } else {
#ifdef skip8k
        if (subwarp.thread_rank() == 0 && ggraph->getDegree(job.node_id) < 8000)
#else
        if (subwarp.thread_rank() == 0)
#endif  // skip8k
        {
          result.AddHighDegree(current_itr, job.node_id);
        }
      }
      subwarp.sync();
      if (subwarp.thread_rank() == 0) job = result.requireOneJob(current_itr);
      job.idx = subwarp.shfl(job.idx, 0);
      job.val = subwarp.shfl(job.val, 0);
      job.node_id = subwarp.shfl(job.node_id, 0);
      subwarp.sync();
    }

    // warp process
    warp.sync();
    // __syncwarp(FULL_WARP_MASK);
    if (warp.thread_rank() == 0)
      job = result.requireOneMidDegreeJob(current_itr);
    warp.sync();
    job.idx = warp.shfl(job.idx, 0);
    job.val = warp.shfl(job.val, 0);
    job.node_id = warp.shfl(job.node_id, 0);
    warp.sync();
    while (job.val) {
      SampleWarpCentic(result, ggraph, state, current_itr, job.idx, job.node_id,
                       buffer);
      warp.sync();
      if (warp.thread_rank() == 0)
        job = result.requireOneMidDegreeJob(current_itr);
      job.idx = warp.shfl(job.idx, 0);
      job.val = warp.shfl(job.val, 0);
      job.node_id = warp.shfl(job.node_id, 0);
    }

    // block process
    __syncthreads();  // cannot reach？？
    __shared__ sample_job high_degree_job;
    if (LTID == 0) {
      job = result.requireOneHighDegreeJob(current_itr);
      high_degree_job.val = job.val;
      high_degree_job.node_id = job.node_id;
    }
    __syncthreads();
    while (high_degree_job.val) {
      SampleBlockCentic(result, ggraph, state, current_itr,
                        high_degree_job.node_id, buffer,
                        vector_packs);  // buffer_pointer
      __syncthreads();
      if (LTID == 0) {
        job = result.requireOneHighDegreeJob(current_itr);
        high_degree_job.val = job.val;
        high_degree_job.node_id = job.node_id;
      }
      __syncthreads();
    }
    __syncthreads();
    if (threadIdx.x == 0) {
      result.NextItr(current_itr);
    }
    __syncthreads();
  }
}

static __global__ void print_result(Sampler *sampler) {
  sampler->result.PrintResult();
}

// void Start_high_degree(Sampler sampler)
float OnlineGBSampleTWC(Sampler &sampler) {
  // orkut max degree 932101

  LOG("%s\n", __FUNCTION__);
#ifdef skip8k
  LOG("skipping 8k\n");
#endif  // skip8k
  // paster(
  //     sizeof(alias_table_constructor_shmem<uint,
  //                                          thread_block_tile<SUBWARP_SIZE>>)
  //                                          *
  //     BLOCK_SIZE / SUBWARP_SIZE);
  // paster(sizeof(warp_table) * WARP_PER_BLK);

  int device;
  hipDeviceProp_t prop;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  int n_sm = prop.multiProcessorCount;

  Sampler *sampler_ptr;
  hipMalloc(&sampler_ptr, sizeof(Sampler));
  CUDA_RT_CALL(hipMemcpy(sampler_ptr, &sampler, sizeof(Sampler),
                          hipMemcpyHostToDevice));
  double start_time, total_time;
  init_kernel_ptr<<<1, 32, 0, 0>>>(sampler_ptr, true);

  // allocate global buffer
  int block_num = n_sm * FLAGS_m;
  int gbuff_size = sampler.ggraph.MaxDegree;

  LOG("alllocate GMEM buffer %d MB\n",
      block_num * gbuff_size * MEM_PER_ELE / 1024 / 1024);

  Vector_pack<uint> *vector_pack_h = new Vector_pack<uint>[block_num];
  for (size_t i = 0; i < block_num; i++) {
    vector_pack_h[i].Allocate(gbuff_size, sampler.device_id);
  }
  CUDA_RT_CALL(hipDeviceSynchronize());
#pragma omp barrier
  Vector_pack<uint> *vector_packs;
  CUDA_RT_CALL(
      hipMalloc(&vector_packs, sizeof(Vector_pack<uint>) * block_num));
  CUDA_RT_CALL(hipMemcpy(vector_packs, vector_pack_h,
                          sizeof(Vector_pack<uint>) * block_num,
                          hipMemcpyHostToDevice));

  //  Global_buffer
  CUDA_RT_CALL(hipDeviceSynchronize());
  start_time = wtime();
  if (FLAGS_debug)
    sample_kernel<<<1, BLOCK_SIZE, 0, 0>>>(sampler_ptr, vector_packs);
  else
    sample_kernel<<<block_num, BLOCK_SIZE, 0, 0>>>(sampler_ptr, vector_packs);

  CUDA_RT_CALL(hipDeviceSynchronize());
  // CUDA_RT_CALL(hipPeekAtLastError());
  total_time = wtime() - start_time;
  LOG("Device %d sampling time:\t%.2f ms ratio:\t %.1f MSEPS\n",
      omp_get_thread_num(), total_time * 1000,
      static_cast<float>(sampler.result.GetSampledNumber() / total_time /
                         1000000));
  sampler.sampled_edges = sampler.result.GetSampledNumber();
  LOG("sampled_edges %d\n", sampler.sampled_edges);
  if (FLAGS_printresult) print_result<<<1, 32, 0, 0>>>(sampler_ptr);
  CUDA_RT_CALL(hipDeviceSynchronize());
  return total_time;
}
