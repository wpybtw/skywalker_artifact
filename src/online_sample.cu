#include "hip/hip_runtime.h"
#include "app.cuh"

static __device__ void SampleWarpCentic(sample_result &result,
                                        gpu_graph *ggraph, hiprandState state,
                                        int current_itr, int idx, int node_id,
                                        void *buffer) {
  alias_table_constructor_shmem<uint, thread_block_tile<32>> *tables =
      (alias_table_constructor_shmem<uint, thread_block_tile<32>> *)buffer;
  alias_table_constructor_shmem<uint, thread_block_tile<32>> *table =
      &tables[WID];
  bool not_all_zero =
      table->loadFromGraph(ggraph->getNeighborPtr(node_id), ggraph,
                           ggraph->getDegree(node_id), current_itr, node_id);
  if (not_all_zero) {
    table->construct();
    table->roll_atomic(&state, result);
  }
  table->Clean();
}

static __device__ void SampleBlockCentic(sample_result &result,
                                         gpu_graph *ggraph, hiprandState state,
                                         int current_itr, int node_id,
                                         void *buffer,
                                         Vector_pack<uint> *vector_packs) {
  alias_table_constructor_shmem<uint, thread_block, BufferType::GMEM> *tables =
      (alias_table_constructor_shmem<uint, thread_block, BufferType::GMEM> *)
          buffer;
  alias_table_constructor_shmem<uint, thread_block, BufferType::GMEM> *table =
      &tables[0];
  table->loadGlobalBuffer(vector_packs);
  __syncthreads();
  bool not_all_zero =
      table->loadFromGraph(ggraph->getNeighborPtr(node_id), ggraph,
                           ggraph->getDegree(node_id), current_itr, node_id);
  __syncthreads();
  if (not_all_zero) {
    table->constructBC();
    uint target_size =
        MIN(ggraph->getDegree(node_id), result.hops[current_itr + 1]);
    table->roll_atomic(target_size, &state, result);
  }
  __syncthreads();
  table->Clean();
}

__global__ void sample_kernel(Sampler *sampler,
                              Vector_pack<uint> *vector_pack) {
  sample_result &result = sampler->result;
  gpu_graph *ggraph = &sampler->ggraph;
  Vector_pack<uint> *vector_packs = &vector_pack[BID];
  __shared__ alias_table_constructor_shmem<uint, thread_block_tile<32>>
      table[WARP_PER_BLK];
  void *buffer = &table[0];
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  __shared__ uint current_itr;
  if (threadIdx.x == 0) current_itr = 0;
  __syncthreads();
  for (; current_itr < result.hop_num - 1;)  // for 2-hop, hop_num=3
  {
    // Vector_gmem<uint> *high_degrees =
    //     &sampler->result.high_degrees[current_itr];
    sample_job job;
    __threadfence_block();
    if (LID == 0) job = result.requireOneJob(current_itr);
    __syncwarp(FULL_WARP_MASK);
    job.idx = __shfl_sync(FULL_WARP_MASK, job.idx, 0);
    job.val = __shfl_sync(FULL_WARP_MASK, job.val, 0);
    job.node_id = __shfl_sync(FULL_WARP_MASK, job.node_id, 0);
    __syncwarp(FULL_WARP_MASK);
    while (job.val) {
      if (ggraph->getDegree(job.node_id) < ELE_PER_WARP) {
        SampleWarpCentic(result, ggraph, state, current_itr, job.idx,
                         job.node_id, buffer);
      } else {
#ifdef skip8k
        if (LID == 0 && ggraph->getDegree(job.node_id) < 8000)
#else
        if (LID == 0)
#endif  // skip8k
          result.AddHighDegree(current_itr, job.node_id);
      }
      __syncwarp(FULL_WARP_MASK);
      if (LID == 0) job = result.requireOneJob(current_itr);
      job.idx = __shfl_sync(FULL_WARP_MASK, job.idx, 0);
      job.val = __shfl_sync(FULL_WARP_MASK, job.val, 0);
      job.node_id = __shfl_sync(FULL_WARP_MASK, job.node_id, 0);
    }
    __syncthreads();
    __shared__ sample_job high_degree_job;
    if (LTID == 0) {
      job = result.requireOneHighDegreeJob(current_itr);
      high_degree_job.val = job.val;
      high_degree_job.node_id = job.node_id;
    }
    __syncthreads();
    while (high_degree_job.val) {
      SampleBlockCentic(result, ggraph, state, current_itr,
                        high_degree_job.node_id, buffer,
                        vector_packs);  // buffer_pointer
      __syncthreads();
      if (LTID == 0) {
        job = result.requireOneHighDegreeJob(current_itr);
        high_degree_job.val = job.val;
        high_degree_job.node_id = job.node_id;
      }
      __syncthreads();
    }
    __syncthreads();
    if (threadIdx.x == 0) {
      // while (!result.checkFinish(current_itr))
      // {
      //   printf("waiting ");
      // }
      result.NextItr(current_itr);
    }
    __syncthreads();
  }
}

static __global__ void print_result(Sampler *sampler) {
  sampler->result.PrintResult();
}

// void Start_high_degree(Sampler sampler)
float OnlineGBSample(Sampler &sampler) {
  // orkut max degree 932101

  LOG("%s\n", __FUNCTION__);
#ifdef skip8k
  LOG("skipping 8k\n");
#endif  // skip8k

  int device;
  hipDeviceProp_t prop;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  int n_sm = prop.multiProcessorCount;

  Sampler *sampler_ptr;
  hipMalloc(&sampler_ptr, sizeof(Sampler));
  CUDA_RT_CALL(hipMemcpy(sampler_ptr, &sampler, sizeof(Sampler),
                          hipMemcpyHostToDevice));
  double start_time, total_time;
  init_kernel_ptr<<<1, 32, 0, 0>>>(sampler_ptr, true);

  // allocate global buffer
  int block_num = n_sm * FLAGS_m;
  int gbuff_size = sampler.ggraph.MaxDegree;

  LOG("alllocate GMEM buffer %d MB\n",
      block_num * gbuff_size * MEM_PER_ELE / 1024 / 1024);

  Vector_pack<uint> *vector_pack_h = new Vector_pack<uint>[block_num];
  for (size_t i = 0; i < block_num; i++) {
    vector_pack_h[i].Allocate(gbuff_size, sampler.device_id);
  }
  CUDA_RT_CALL(hipDeviceSynchronize());
#pragma omp barrier
  Vector_pack<uint> *vector_packs;
  CUDA_RT_CALL(
      hipMalloc(&vector_packs, sizeof(Vector_pack<uint>) * block_num));
  CUDA_RT_CALL(hipMemcpy(vector_packs, vector_pack_h,
                          sizeof(Vector_pack<uint>) * block_num,
                          hipMemcpyHostToDevice));

  //  Global_buffer
  CUDA_RT_CALL(hipDeviceSynchronize());
  start_time = wtime();
#ifdef check
  sample_kernel<<<1, BLOCK_SIZE, 0, 0>>>(sampler_ptr, vector_packs);
#else
  sample_kernel<<<block_num, BLOCK_SIZE, 0, 0>>>(sampler_ptr, vector_packs);
#endif
  CUDA_RT_CALL(hipDeviceSynchronize());
  // CUDA_RT_CALL(hipPeekAtLastError());
  total_time = wtime() - start_time;
  LOG("Device %d sampling time:\t%.2f ms ratio:\t %.1f MSEPS\n",
      omp_get_thread_num(), total_time * 1000,
      static_cast<float>(sampler.result.GetSampledNumber() / total_time /
                         1000000));
  sampler.sampled_edges = sampler.result.GetSampledNumber();
  LOG("sampled_edges %d\n", sampler.sampled_edges);
  if (FLAGS_printresult) print_result<<<1, 32, 0, 0>>>(sampler_ptr);
  CUDA_RT_CALL(hipDeviceSynchronize());
  return total_time;
}
