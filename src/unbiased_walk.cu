#include "hip/hip_runtime.h"
/*
 * @Description: just perform RW
 * @Date: 2020-11-30 14:30:06
 * @LastEditors: PengyuWang
 * @LastEditTime: 2021-01-10 15:09:28
 * @FilePath: /skywalker/src/unbiased_walk.cu
 */
#include "app.cuh"

#define PV 2.0f
#define QV 0.5f
#define MAX_SCALE MAX(PV, QV)

__global__ void Node2vecKernelStaticBuffer(Walker *walker) {
  Jobs_result<JobType::RW, uint> &result = walker->result;
  gpu_graph *graph = &walker->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);
  __shared__ matrixBuffer<BLOCK_SIZE, 31, uint> buffer;
  buffer.Init();
  size_t idx_i = TID;
  uint lastV = idx_i;
  if (idx_i < result.size) {
    result.length[idx_i] = result.hop_num - 1;
    for (uint current_itr = 0; current_itr < result.hop_num - 1;
         current_itr++) {
      uint src_id = result.GetData(current_itr, idx_i);
      uint src_degree = graph->getDegree((uint)src_id);
      if (src_degree == 0) {
        result.length[idx_i] = current_itr;
        buffer.Finish();
        return;
      } else if (src_degree > 1) {
        uint outV;
        do {
          uint x = (int)floor(hiprand_uniform(&state) * src_degree);
          uint y = (int)floor(hiprand_uniform(&state) * MAX_SCALE);
          float h;
          outV = graph->getOutNode(src_id, x);
          if (graph->CheckConnect(lastV, outV)) {
            h = QV;
          } else if (lastV == outV) {
            h = PV;
          } else {
            h = 1.0;
          }
          if (y < h) break;
        } while (true);
        buffer.Set(outV);
      } else {
        buffer.Set(graph->getOutNode(src_id, 0));
      }
      lastV = src_id;
      buffer.CheckFlush(result.data + result.hop_num * idx_i, current_itr);
    }
    buffer.Flush(result.data + result.hop_num * idx_i, 0);
  }
}
__global__ void UnbiasedWalkKernelStaticBuffer(Walker *walker, float *tp) {
  Jobs_result<JobType::RW, uint> &result = walker->result;
  gpu_graph *graph = &walker->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);
  __shared__ matrixBuffer<BLOCK_SIZE, 31, uint> buffer;
  buffer.Init();
  size_t idx_i = TID;
  if (idx_i < result.size) {
    result.length[idx_i] = result.hop_num - 1;
    uint src_id;
    bool alive = true;
    for (uint current_itr = 0; current_itr < result.hop_num - 1;
         current_itr++) {
      if (alive) {
        src_id =
            (current_itr == 0) ? result.GetData(current_itr, idx_i) : src_id;
        uint src_degree = graph->getDegree((uint)src_id);
        if (src_degree == 0 || hiprand_uniform(&state) < *tp) {
          result.length[idx_i] = current_itr;
          // buffer.Finish();
          alive = false;
        } else if (src_degree > 1) {
          uint candidate = (int)floor(hiprand_uniform(&state) * src_degree);
          uint next_src = graph->getOutNode(src_id, candidate);
          buffer.Set(next_src);
          src_id = next_src;
        } else {
          uint next_src = graph->getOutNode(src_id, 0);
          buffer.Set(next_src);
          src_id = next_src;
        }
      }
      buffer.CheckFlush(result.data + result.hop_num * idx_i, current_itr);
    }
    buffer.Flush(result.data + result.hop_num * idx_i, 0);
  }
}
__global__ void UnbiasedWalkKernelStatic(Walker *walker, float *tp) {
  Jobs_result<JobType::RW, uint> &result = walker->result;
  gpu_graph *graph = &walker->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  size_t idx_i = TID;
  if (idx_i < result.size) {
    result.length[idx_i] = result.hop_num - 1;
    for (uint current_itr = 0; current_itr < result.hop_num - 1;
         current_itr++) {
      uint src_id = result.GetData(current_itr, idx_i);
      uint src_degree = graph->getDegree((uint)src_id);
      if (src_degree == 0 || hiprand_uniform(&state) < *tp) {
        result.length[idx_i] = current_itr;
        break;
      } else if (1 < src_degree) {
        uint candidate = (int)floor(hiprand_uniform(&state) * src_degree);
        *result.GetDataPtr(current_itr + 1, idx_i) =
            graph->getOutNode(src_id, candidate);
      } else {
        *result.GetDataPtr(current_itr + 1, idx_i) =
            graph->getOutNode(src_id, 0);
      }
    }
  }
}
__global__ void UnbiasedWalkKernel(Walker *walker, float *tp) {
  Jobs_result<JobType::RW, uint> &result = walker->result;
  gpu_graph *graph = &walker->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);
  for (size_t idx_i = TID; idx_i < result.size;
       idx_i += gridDim.x * blockDim.x) {
    result.length[idx_i] = result.hop_num - 1;
    for (uint current_itr = 0; current_itr < result.hop_num - 1;
         current_itr++) {
      uint src_id = result.GetData(current_itr, idx_i);
      uint src_degree = graph->getDegree((uint)src_id);
      // if(idx_i==0) printf("src_id %d src_degree %d\n",src_id,src_degree );
      if (src_degree == 0 || hiprand_uniform(&state) < *tp) {
        result.length[idx_i] = current_itr;
        break;
      } else if (1 < src_degree) {
        uint candidate = (int)floor(hiprand_uniform(&state) * src_degree);
        *result.GetDataPtr(current_itr + 1, idx_i) =
            graph->getOutNode(src_id, candidate);
      } else {
        *result.GetDataPtr(current_itr + 1, idx_i) =
            graph->getOutNode(src_id, 0);
      }
    }
  }
}
__global__ void UnbiasedWalkKernelPerItr(Walker *walker, uint current_itr) {
  Jobs_result<JobType::RW, uint> &result = walker->result;
  gpu_graph *graph = &walker->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);
  // for (uint current_itr = 0; current_itr < result.hop_num - 1;
  // current_itr++)
  // {
  if (TID < result.frontier.Size(current_itr)) {
    size_t idx_i = result.frontier.Get(current_itr, TID);
    uint src_id = result.GetData(current_itr, idx_i);
    uint src_degree = graph->getDegree((uint)src_id);
    result.length[idx_i] = current_itr;
    if (1 < src_degree) {
      int col = (int)floor(hiprand_uniform(&state) * src_degree);
      uint candidate = col;
      *result.GetDataPtr(current_itr + 1, idx_i) =
          graph->getOutNode(src_id, candidate);
      result.frontier.SetActive(current_itr + 1, idx_i);
    } else if (src_degree == 1) {
      *result.GetDataPtr(current_itr + 1, idx_i) = graph->getOutNode(src_id, 0);
      result.frontier.SetActive(current_itr + 1, idx_i);
    }
  }
}

__global__ void Reset(Walker *walker, uint current_itr) {
  if (TID == 0) walker->result.frontier.Reset(current_itr);
}
__global__ void GetSize(Walker *walker, uint current_itr, uint *size) {
  if (TID == 0) *size = walker->result.frontier.Size(current_itr);
}
static __global__ void print_result(Walker *walker) {
  walker->result.PrintResult();
}

float UnbiasedWalk(Walker &walker) {
  LOG("%s\n", __FUNCTION__);
  int device;
  hipDeviceProp_t prop;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  int n_sm = prop.multiProcessorCount;

  Walker *sampler_ptr;
  hipMalloc(&sampler_ptr, sizeof(Walker));
  CUDA_RT_CALL(
      hipMemcpy(sampler_ptr, &walker, sizeof(Walker), hipMemcpyHostToDevice));

  float *tp_d, tp;
  tp = FLAGS_tp;
  hipMalloc(&tp_d, sizeof(float));
  CUDA_RT_CALL(hipMemcpy(tp_d, &tp, sizeof(float), hipMemcpyHostToDevice));

  double start_time, total_time;
  // init_kernel_ptr<<<1, 32, 0, 0>>>(sampler_ptr);

  // hipEvent_t start, stop;
  // hipEventCreate(&start);
  // hipEventCreate(&stop);

  // allocate global buffer
  int block_num = n_sm * FLAGS_m;
  CUDA_RT_CALL(hipDeviceSynchronize());
  CUDA_RT_CALL(hipPeekAtLastError());

  uint size_h, *size_d;
  hipMalloc(&size_d, sizeof(uint));

  // hipEventRecord(start);
  start_time = wtime();

  if (FLAGS_node2vec) {
    Node2vecKernelStaticBuffer<<<walker.num_seed / BLOCK_SIZE + 1, BLOCK_SIZE,
                                 0, 0>>>(sampler_ptr);
  } else if (!FLAGS_peritr) {
    if (FLAGS_static) {
      if (FLAGS_buffer)
        UnbiasedWalkKernelStaticBuffer<<<walker.num_seed / BLOCK_SIZE + 1,
                                         BLOCK_SIZE, 0, 0>>>(sampler_ptr, tp_d);
      else
        UnbiasedWalkKernelStatic<<<walker.num_seed / BLOCK_SIZE + 1, BLOCK_SIZE,
                                   0, 0>>>(sampler_ptr, tp_d);
    } else
      UnbiasedWalkKernel<<<block_num, BLOCK_SIZE, 0, 0>>>(sampler_ptr, tp_d);
  } else {
    for (uint current_itr = 0; current_itr < walker.result.hop_num - 1;
         current_itr++) {
      GetSize<<<1, 32, 0, 0>>>(sampler_ptr, current_itr, size_d);
      CUDA_RT_CALL(
          hipMemcpy(&size_h, size_d, sizeof(uint), hipMemcpyDeviceToHost));
      if (size_h > 0) {
        UnbiasedWalkKernelPerItr<<<size_h / BLOCK_SIZE + 1, BLOCK_SIZE, 0, 0>>>(
            sampler_ptr, current_itr);
        Reset<<<1, 32, 0, 0>>>(sampler_ptr, current_itr);
      } else {
        break;
      }
    }
  }

  CUDA_RT_CALL(hipDeviceSynchronize());
  // hipEventRecord(stop);
  // hipEventSynchronize(stop);

  // CUDA_RT_CALL(hipPeekAtLastError());
  total_time = wtime() - start_time;
  // float milliseconds = 0;
  // hipEventElapsedTime(&milliseconds, start, stop);
  // printf("cuda event time %f \n",milliseconds);
  LOG("Device %d sampling time:\t%.6f ratio:\t %.2f MSEPS sampled %u\n",
      omp_get_thread_num(), total_time,
      static_cast<float>(walker.result.GetSampledNumber() / total_time /
                         1000000),
      walker.result.GetSampledNumber());
  walker.sampled_edges = walker.result.GetSampledNumber();
  if (FLAGS_printresult) print_result<<<1, 32, 0, 0>>>(sampler_ptr);
  CUDA_RT_CALL(hipDeviceSynchronize());
  return total_time;
}
