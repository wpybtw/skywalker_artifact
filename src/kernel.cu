#include "hip/hip_runtime.h"
/*
 * @Description:
 * @Date: 2020-11-25 13:28:14
 * @LastEditors: PengyuWang
 * @LastEditTime: 2020-12-07 16:32:47
 * @FilePath: /sampling/src/kernel.cu
 */
#include "gpu_graph.cuh"
#include "kernel.cuh"

// __global__ void initSeed(ResultBase<uint> *results, uint *seeds, size_t size)
// {
//   if (TID < size) {
//     results[TID].data[0] = seeds[TID];
//   }
// }
__global__ void BindResultKernel(Walker *walker) {
  if (TID == 0) walker->BindResult();
}

__global__ void init_kernel_ptr(Sampler *sampler, bool biasInit) {
  if (TID == 0) {
    sampler->result.setAddrOffset();
    if (biasInit)
      for (size_t i = 0; i < sampler->result.hop_num; i++) {
        sampler->result.high_degrees[i].Init();
      }
  }
}

__global__ void init_kernel_ptr(Walker *sampler, bool biasInit) {
  if (TID == 0) {
    sampler->result.setAddrOffset();
    if (biasInit)
      for (size_t i = 0; i < sampler->result.hop_num; i++) {
        sampler->result.high_degrees[i].Init();
      }
  }
}

__device__ bool AddTillSize(uint *size,
                            size_t target_size)  // T *array,       T t,
{
  uint old = atomicAdd(size, 1);
  if (old < target_size) {
    return true;
  }
  return false;
}